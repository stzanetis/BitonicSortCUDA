#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include "../Utilities/utilities.cuh"

__global__ void bitonicSortKernel(int *dev_values, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    for (unsigned int k = 2; k <= N; k <<= 1) {
        for (unsigned int j = k >> 1; j > 0; j >>= 1) {
            unsigned int ixj = i ^ j;

            if (ixj > i) {
                if ((i & k) == 0) {
                    if (dev_values[i] > dev_values[ixj]) {
                        int temp = dev_values[i];
                        dev_values[i] = dev_values[ixj];
                        dev_values[ixj] = temp;
                    }
                } else {
                    if (dev_values[i] < dev_values[ixj]) {
                        int temp = dev_values[i];
                        dev_values[i] = dev_values[ixj];
                        dev_values[ixj] = temp;
                    }
                }
            }
            __syncthreads();
        }
    }
}

void bitonicSort(int *values, int N) {
    int *dev_values;
    size_t size = N * sizeof(int);

    hipMalloc((void**)&dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    dim3 blocks((N + 511) / 512);
    dim3 threads(512);

    bitonicSortKernel<<<blocks, threads>>>(dev_values, N);
    hipDeviceSynchronize();

    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " {q} {p}" << std::endl;
        return 1;
    } else {
        int p = std::atoi(argv[1]);
        int q = std::atoi(argv[2]);
    }

    int N = 1 << (q + p);   // N = 2^(q + p)
    int *values = new int[N];
    std::cout << "Number of elements: " << N << std::endl;

    generateArray(values, N);

    auto start = std::chrono::high_resolution_clock::now();
    bitonicSort(values, N);
    auto end = std::chrono::high_resolution_clock::now();

    if (isSorted(values, N)) {
        std::cout << "The array is sorted correctly." << std::endl;
    } else {
        std::cout << "The array is NOT sorted correctly." << std::endl;
    }

    std::chrono::duration<double> duration = end - start;
    std::cout << "V1 Bitonic sort took " << duration.count() << " seconds." << std::endl;

    delete[] values;
    return 0;
}