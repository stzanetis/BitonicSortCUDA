#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include "../Utilities/utilities.h"

__global__ void bitonicSortStep(int *dev_values, int threads, int stage, int step) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int partner = tid ^ step;
    
    if (tid < partner) {
        if (!(tid & stage)) {
            if (dev_values[tid] > dev_values[partner]) {
                int temp = dev_values[tid];
                dev_values[tid] = dev_values[partner];
                dev_values[partner] = temp;
            }
        } else {
            if (dev_values[tid] < dev_values[partner]) {
                int temp = dev_values[tid];
                dev_values[tid] = dev_values[partner];
                dev_values[partner] = temp;
            }
        }
    } else {
        tid += threads;
        partner += threads;

        if (!(tid & stage)) {
            if (dev_values[tid] < dev_values[partner]) {
                int temp = dev_values[tid];
                dev_values[tid] = dev_values[partner];
                dev_values[partner] = temp;
            }
        } else {
            if (dev_values[tid] > dev_values[partner]) {
                int temp = dev_values[tid];
                dev_values[tid] = dev_values[partner];
                dev_values[partner] = temp;
            }
        }
    }
}

__global__ void localSort(int *dev_values, int N, int stage, int step) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int offset = N >> 1;

    if (tid < offset) {
        do {
            while (step > 0) {
                unsigned int partner = tid ^ step;
                if (partner > tid) {
                    if (!(tid & stage)) {
                        if (dev_values[tid] > dev_values[partner]) {
                            int temp = dev_values[tid];
                            dev_values[tid] = dev_values[partner];
                            dev_values[partner] = temp;
                        }
                    } else {
                        if (dev_values[tid] < dev_values[partner]) {
                            int temp = dev_values[tid];
                            dev_values[tid] = dev_values[partner];
                            dev_values[partner] = temp;
                        }
                    }
                } else {
                    tid += offset;
                    partner += offset;

                    if (!(tid & stage)) {
                        if (dev_values[tid] < dev_values[partner]) {
                            int temp = dev_values[tid];
                            dev_values[tid] = dev_values[partner];
                            dev_values[partner] = temp;
                        }
                    } else {
                        if (dev_values[tid] > dev_values[partner]) {
                            int temp = dev_values[tid];
                            dev_values[tid] = dev_values[partner];
                            dev_values[partner] = temp;
                        }
                    }
                    tid -= offset;
                }
                step >>= 1;
                __syncthreads();
            }
            stage <<= 1;
            step = stage >> 1;
        } while (stage <= min(N, 1024));
    }
}

void bitonicSort(int *values, int N) {
    int *dev_values;
    size_t size = N * sizeof(int);
    int threads_per_block = 1024;
    int blocks = ((N/2) - 1) / threads_per_block + 1;

    hipMalloc((void**)&dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    localSort<<<blocks, threads_per_block>>>(dev_values, N, 2, 1);

    for (int stage = 2048; stage <= N; stage <<= 1) {
        for (int step = stage >> 1; step > 512; step >>= 1) {
            bitonicSortStep<<<blocks, threads_per_block>>>(dev_values, (N/2), stage, step);
            hipDeviceSynchronize();
        }
        localSort<<<blocks, threads_per_block>>>(dev_values, N, stage, 512);
    }    

    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}

int main(int argc, char *argv[]) {
    int N;
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <n>" << std::endl;
        return 1;
    } else {
        N = 1 << (std::atoi(argv[1]));
    }

    int *values = new int[N];
    std::cout << "Number of elements to be sorted: " << N << std::endl;
    
    generateArray(values, N);

    //printf("Unsorted array: ");
    //printArray(values, N);

    auto start = std::chrono::high_resolution_clock::now();
    bitonicSort(values, N);
    auto end = std::chrono::high_resolution_clock::now();

    //printf("Sorted array: ");
    //printArray(values, N);

    //if (isSorted(values, N)) {
    //    std::cout << "The array is sorted correctly." << std::endl;
    //} else {
    //    std::cout << "The array is NOT sorted correctly." << std::endl;
    //}

    std::chrono::duration<double> duration = end - start;
    std::cout << "V1 Bitonic sort took " << duration.count() << " seconds." << std::endl;

    delete[] values;
    return 0;
}